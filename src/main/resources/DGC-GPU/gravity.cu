#include "hip/hip_runtime.h"
int* h_numberInstancesClass;
__constant__ char ISNUMERICAL[256];

__device__ float distance(int instanciaA, int instanciaB, float* d_weights, int numAttributes, float* instancesData, int numberInstances_A, int base)
{
	float distance = 0.0f;

	for(int i = 0; i < numAttributes; i++)
		if(ISNUMERICAL[i])
		{
			float diff = instancesData[instanciaB + i*numberInstances_A] - instancesData[instanciaA + i*numberInstances_A];
			distance += d_weights[base + i] * diff * diff;
		}
		else
			distance += d_weights[base + i] * (instancesData[instanciaB + i*numberInstances_A] == instancesData[instanciaA + i*numberInstances_A] ? 0 : 1);

	return distance;
}

__global__ void kernelCalculateGravity(unsigned char* result, int Class, float* gravityValues, float* instancesData, int* instancesClass, int numberAttributes, int numberInstances, int numberInstances_A, int numberClasses, int* numberInstancesClass, float* d_weights) 
{
	int instance = blockDim.y * blockIdx.y + threadIdx.y;

	if(instance < numberInstances)
	{
		float gravityValue = 0.0f;

		for(int i = 0; i < numberInstances; i++)
			if(instancesClass[i] == Class && i != instance)
				gravityValue += 1.0f / distance(instance, i, d_weights, numberAttributes, instancesData, numberInstances_A, blockIdx.x*numberAttributes*numberClasses + instancesClass[i]*numberAttributes);

		gravityValue *= 1.0f - ((numberInstancesClass[Class] - 1) / (float) numberInstances);

		int memPosition = blockIdx.x*numberInstances_A + instance;

		if(gravityValue > gravityValues[memPosition])
		{
			gravityValues[memPosition] = gravityValue;

			if(instancesClass[instance] == Class)
				result[memPosition] = 0; // HIT
			else
				result[memPosition] = 1; // FAIL
		}
	}
}

/**
 * Reduction GPU Confusion Matrix kernel
 */

__global__ void MC_kernelGravity(unsigned char* result, jfloat* fitness, int numberInstances, int numberInstances_A) 
{
	__shared__ int MC[THREADS_EVAL_BLOCK];

	MC[threadIdx.y] = 0;

	int base = blockIdx.x*numberInstances_A + threadIdx.y;
	int top =  numberInstances - threadIdx.y;

	// Performs the reduction of the thread corresponding values
	for(int i = 0; i < top; i+=THREADS_EVAL_BLOCK)
	{
		MC[threadIdx.y] += result[base + i];
	}

	__syncthreads();

	// Calculates the final amount
	if(threadIdx.y == 0)
	{
		int fails = 0;
		
		for(int i = 0; i < THREADS_EVAL_BLOCK; i++)
			fails += MC[i];
		
		// Set the fitness to the individual
		fitness[blockIdx.x] = fails / (float) numberInstances;
	}
}

/**
 * GPU device thread that performs the evaluation of a portion of the population
 *
 * @param The job plan for the thread
 */
CUT_THREADPROC gpuThreadGravity(Plan *plan)
{
	// Set the GPU device number, each thread on a different GPU
	int device = plan->thread;
	hipSetDeviceFlags(hipDeviceScheduleSpin);
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if(deviceCount == 3 && device == 1) device = 2;
	hipSetDevice(device);

    float* h_fitness;
	float* d_gravity;
	float* d_weights;
	float* d_instancesData;
	float* h_weights;
	int *d_instancesClass;
	int* d_numberInstancesClass;
	int threadPopulationSize;
	unsigned char* d_result;
	jfloat* d_fitness;
	JNIEnv* env;
	JavaVM* jvm;

	hipHostMalloc((void**)&h_fitness, BLOCK_SIZE_GRAVITY*sizeof(float));
	hipHostMalloc((void**)&h_weights, numberAttributes*numClasses*BLOCK_SIZE_GRAVITY*sizeof(float));
	
	// GPU dynamic memory allocation
	hipMalloc((void**) &d_numberInstancesClass, numClasses*sizeof(int));	
	hipMalloc((void**) &d_weights, numberAttributes*numClasses*BLOCK_SIZE_GRAVITY*sizeof(jfloat));	
	hipMalloc((void**) &d_fitness, BLOCK_SIZE_GRAVITY*sizeof(jfloat));	
	hipMalloc((void**) &d_instancesData, numberAttributes*numberInstances_A*sizeof(float));
	hipMalloc((void**) &d_instancesClass, numberInstances*sizeof(int));
	hipMalloc((void**) &d_result, BLOCK_SIZE_GRAVITY * numberInstances_A * sizeof(unsigned char));
	hipMalloc((void**) &d_gravity, BLOCK_SIZE_GRAVITY * numberInstances_A * sizeof(float));

	// Copy instances data and classes to the GPU
	hipMemcpy(d_instancesData, h_instancesData, numberAttributes*numberInstances_A*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy(d_instancesClass, h_instancesClass, numberInstances*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy(d_numberInstancesClass, h_numberInstancesClass, numClasses*sizeof(int), hipMemcpyHostToDevice );

	// Signal: thread is ready to evaluate
	SEM_POST(&post_sem[plan->thread]);

	Get_VM(&jvm, &env);

	dim3 threads_evaluate(1, THREADS_EVAL_BLOCK);
	dim3 threads_mc(1,THREADS_EVAL_BLOCK);
	
	bool firstTime = true;
	
	do
	{
		// Wait until evaluation is required
		SEM_WAIT (&wait_sem[plan->thread]);

		if(evaluate)
		{
			// Get the methods from Java
			jclass cls = env->GetObjectClass(algorithm);
		
			jmethodID midR = env->GetMethodID(cls, "getWeight", "(III)F");	
			jmethodID midW = env->GetMethodID(cls, "setFitness", "(IF)V");
			
			if(firstTime)
			{
				jmethodID midN = env->GetMethodID(cls, "isNumerical", "(I)Z");	
				
				for(int i = 0; i < numberAttributes; i++)
				{
					jboolean isNumericalValue = env->CallCharMethod(algorithm, midN, i);
					hipMemcpyToSymbol(HIP_SYMBOL("ISNUMERICAL"), &isNumericalValue, sizeof(jboolean), i*sizeof(jboolean), hipMemcpyHostToDevice);
				}
				
				firstTime = false;
			}
	
			// Calculate the thread population size
			threadPopulationSize = (int)ceil(populationSize/(float)numThreads);

			// If population overflow, recalculate the thread actual population size
			if((plan->thread + 1) * threadPopulationSize > populationSize)
			{
				if((threadPopulationSize = populationSize - threadPopulationSize * plan->thread) < 0)
					threadPopulationSize = 0;
			}
			if(threadPopulationSize > 0)
			{
				// Calculate the base index of the individual for this thread
				int base = plan->thread * (int)ceil(populationSize/(float)numThreads);

				int blockIdxSize = BLOCK_SIZE_GRAVITY;	

				// Population is evaluated using blocks of BLOCK_SIZE_GRAVITY individuals
				for(int j = 0; j < threadPopulationSize; j += BLOCK_SIZE_GRAVITY)
				{
					// If the last block size is smaller, fix the block size to the number of the rest of individuals 
					if(j+BLOCK_SIZE_GRAVITY > threadPopulationSize)
						blockIdxSize = threadPopulationSize - j;
									
					// Copy each individual in the block from the thread population to the GPU
					for(int i = 0; i < blockIdxSize; i++)
						for(int k = 0; k < numClasses; k++)
							for(int l = 0; l < numberAttributes; l++)
								h_weights[i*numberAttributes*numClasses + k*numberAttributes + l] = env->CallFloatMethod(algorithm,midR,base+j+i,l,k);

								hipMemcpy(d_weights, h_weights, numberAttributes*numClasses*blockIdxSize*sizeof(float), hipMemcpyHostToDevice );

								// Setup evaluation grid size	
								dim3 grid_evaluate(blockIdxSize, (int)ceil(numberInstances/(float)THREADS_EVAL_BLOCK));

								hipMemset(d_gravity, 0, BLOCK_SIZE_GRAVITY * numberInstances_A * sizeof(float));

								// Evaluation kernel call					
								for(int i = 0; i < numClasses; i++)
									kernelCalculateGravity <<< grid_evaluate, threads_evaluate >>> (d_result, i, d_gravity, d_instancesData, d_instancesClass, numberAttributes, numberInstances, numberInstances_A, numClasses, d_numberInstancesClass, d_weights);

								// Setup reduction grid size
								dim3 grid_mc(blockIdxSize, 1);

								// Reduction kernel call
								MC_kernelGravity <<< grid_mc, threads_mc >>> (d_result, d_fitness, numberInstances, numberInstances_A);

								// Copy the fitness values from the GPU to Host memory and set them to the individuals
								hipMemcpy(h_fitness, d_fitness, blockIdxSize*sizeof(jfloat), hipMemcpyDeviceToHost );	

								for(int i = 0; i < blockIdxSize; i++)
								{
									env->CallVoidMethod(algorithm, midW, base + j + i, h_fitness[i]);
								}							
				}
			}
		}
		else
		{
			// Algorithm finished, free dynamic memory
			hipFree(d_instancesData);    
			hipFree(d_instancesClass);
			hipFree(d_result);
			hipFree(d_fitness);
			hipFree(d_weights);
			hipFree(d_numberInstancesClass);

			hipHostFree(h_fitness);
			hipHostFree(h_weights);
		}

		// Evaluation finished
		SEM_POST(&post_sem[plan->thread]);

	}while(evaluate);

	jvm->DetachCurrentThread();
	CUT_THREADEND;
}

/**
 * Function executed when nativeFree() call from Java
 */
JNIEXPORT void JNICALL
Java_net_sf_jclec_problem_classification_dgc_DGCEvaluatorGPU_nativeFree(JNIEnv *env, jobject obj)
{
	nativeFree(env,obj);
}

/**
 * Function executed when nativeMalloc() call from Java
 */
JNIEXPORT void JNICALL
Java_net_sf_jclec_problem_classification_dgc_DGCEvaluatorGPU_nativeMalloc(JNIEnv *env, jobject obj, jint popSize, jint jnumThreads, jint jnumberAttributes, jint jnumberInstances, jint jnumClasses, jobject jalgorithm)
{
	algorithm = jalgorithm;
	numThreads = jnumThreads;
	numberAttributes = jnumberAttributes;
	numClasses = jnumClasses;
	numberInstances = jnumberInstances;
	numberInstances_A = ceil(numberInstances/(float)ALIGNMENT)*ALIGNMENT;

	// Set up semaphores
	for(int i = 0; i < numThreads; i++)
	{
		SEM_INIT (&wait_sem[i], 0);
		SEM_INIT (&post_sem[i], 0);
	}

	jclass cls = env->GetObjectClass(algorithm);
	jmethodID midR = env->GetMethodID(cls, "getValue", "(II)F");
	jmethodID midRR = env->GetMethodID(cls, "getNumberInstances", "(I)I");
	jmethodID midRRR = env->GetMethodID(cls, "getClassValue", "(I)F");

	h_instancesData = (float*)malloc(numberAttributes*numberInstances_A*sizeof(float));
	h_instancesClass = (int*)malloc(numberInstances*sizeof(int));	
	h_numberInstancesClass = (int*)malloc(numClasses*sizeof(int));

	for(int i = 0; i < numClasses; i++)
	{
		h_numberInstancesClass[i] = env->CallIntMethod(algorithm,midRR,i);
	}

	// Copy dataset data from Java
	for(int i = 0; i < numberInstances; i++)
	{
		for(int j = 0; j < numberAttributes; j++)
			h_instancesData[j*numberInstances_A+i] = env->CallFloatMethod(algorithm,midR,i,j);

			h_instancesClass[i] = (int) env->CallFloatMethod(algorithm,midRRR,i);
	}

	// Set up threads plans
	for(int i = 0; i < numThreads; i++)
	{
		plan[i].thread = i;
		plan[i].size = (int)ceil(popSize/(float)numThreads);
	}

	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	if(numThreads > deviceCount)
	{
		fprintf(stderr, "Can't use %d threads. CUDA devices (non-display) count is %d\n",numThreads, deviceCount);
		exit(0);
	}

	for(int i = 0; i < numThreads; i++)
		threadID[i] = cutStartThread((CUT_THREADROUTINE)gpuThreadGravity, (void *)&plan[i]);

	// SIGNAL: threads ready to evaluate
	for(int i = 0; i < numThreads; i++)
		SEM_WAIT (&post_sem[i]);
}

/**
 * Function executed when nativeEvaluate() call from Java
 * 
 * @param The number of individuals, the actual class to classify and the algorithm
 */
JNIEXPORT void JNICALL
Java_net_sf_jclec_problem_classification_dgc_DGCEvaluatorGPU_nativeEvaluate(JNIEnv *env, jobject obj, jint size, jobject jalgorithm)
{
	evaluate = true;

	algorithm = jalgorithm;
	populationSize = size;

	// SIGNAL: wake up threads to evaluate
	for(int i = 0; i < numThreads && i < size; i++)
		SEM_POST (&wait_sem[i]);

	// Wait until threads finish
	for(int i = 0; i < numThreads && i < size; i++)
		SEM_WAIT (&post_sem[i]);
}
